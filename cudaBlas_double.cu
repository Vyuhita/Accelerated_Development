#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Function to initialize a matrix with random values
void initializeMatrixDouble(double* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            matrix[i * size + j] = 1.0; // Assuming initialization with 1.0 for simplicity
        }
    }
}

// Function to print a matrix (for debugging purposes)
void printMatrixDouble(const double* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            std::cout << matrix[i * size + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    // Matrix size (assuming square matrices)
    const int matrixSize = 16384;  // Adjust the size based on your requirements

    // Allocate host memory for matrices
    double* h_A = new double[matrixSize * matrixSize];
    double* h_B = new double[matrixSize * matrixSize];
    double* h_C = new double[matrixSize * matrixSize];

    // Initialize matrices with random values
    initializeMatrixDouble(h_A, matrixSize);
    initializeMatrixDouble(h_B, matrixSize);

    // Allocate device memory for matrices
    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, matrixSize * matrixSize * sizeof(double));
    hipMalloc((void**)&d_B, matrixSize * matrixSize * sizeof(double));
    hipMalloc((void**)&d_C, matrixSize * matrixSize * sizeof(double));

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, matrixSize * matrixSize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixSize * matrixSize * sizeof(double), hipMemcpyHostToDevice);

    // Initialize cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start the timer for matrix multiplication
    hipEventRecord(start);

    // Perform matrix multiplication using cuBLAS
    const double alpha = 1.0;
    const double beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrixSize, matrixSize, matrixSize, &alpha, d_A, matrixSize, d_B, matrixSize, &beta, d_C, matrixSize);

    // Stop the timer for matrix multiplication
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Elapsed time for matrix multiplication: " << elapsedTime / 1000.0 << " seconds" << std::endl;

    // Copy result matrix from device to host
    hipMemcpy(h_C, d_C, matrixSize * matrixSize * sizeof(double), hipMemcpyDeviceToHost);

    // Print the result matrix if needed
    printMatrixDouble(h_C, matrixSize);

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}

