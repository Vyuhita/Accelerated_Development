#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define N 4096
#define BLOCK_SIZE 32
#define NUM_GPUS 2

// Function to print a matrix
void printMatrix(float* matrix, int size) {
    std::cout << std::fixed << std::setprecision(2);	
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            std::cout << std::setw(8) << matrix[i * size + j] << " ";
        }
        std::cout << std::endl;
    }
}

// Define the timer function
float timer(hipEvent_t start, hipEvent_t stop) {
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    return elapsedTime;
}

__global__ void Kernel_A(float* a, float* b, float* c, int matrixSize) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * BLOCK_SIZE + ty;
    int col = bx * BLOCK_SIZE + tx;

    __shared__ float shared_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_b[BLOCK_SIZE][BLOCK_SIZE];

    float sum = 0.0f;

    for (int m = 0; m < matrixSize / BLOCK_SIZE; ++m) {
        shared_a[ty][tx] = a[row * matrixSize + (m * BLOCK_SIZE + tx)];
        shared_b[ty][tx] = b[col + (m * BLOCK_SIZE + ty) * matrixSize];
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            sum += shared_a[ty][k] * shared_b[k][tx];
        }
        __syncthreads();
    }

    c[row * matrixSize + col] = sum;
}

void matrixMultiplication(int device, float* h_a, float* h_b, float* h_c, int numRows) {
    float* d_a;
    float* d_b;
    float* d_c;

    int size = N * numRows * sizeof(float);

    //device memory allocation
    hipSetDevice(device);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Create CUDA streams
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Record start time
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    // Copy data from host to device using streams
    hipMemcpyAsync(d_a, h_a, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, h_b, size, hipMemcpyHostToDevice, stream);

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (numRows + blockDim.y - 1) / blockDim.y);

    // Launch Kernel_A
    Kernel_A<<<gridDim, blockDim, 0, stream>>>(d_a, d_b, d_c, N);

    // Synchronize the stream
    hipStreamSynchronize(stream);

    // Copy result from device to host
    hipMemcpyAsync(h_c, d_c, size, hipMemcpyDeviceToHost, stream);

    // Record end time
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    // Calculate elapsed time including memory transfers and kernel execution
    float elapsedTime = timer(startEvent, stopEvent);
    std::cout << "Elapsed Time on GPU " << device << ": " << elapsedTime << " ms" << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Destroy CUDA stream
    hipStreamDestroy(stream);
}

int main() {
    // Initialize two GPUs
    int numGPUs;
    hipGetDeviceCount(&numGPUs);

    if (numGPUs < NUM_GPUS) {
        std::cerr << "This code requires at least " << NUM_GPUS << " GPUs." << std::endl;
        return 1;
    }

    float gpuTimes[NUM_GPUS] = {0.0};

    // Create data matrices
    float* h_a = (float*)malloc(N * N * sizeof(float));
    float* h_b = (float*)malloc(N * N * sizeof(float));
    float* h_c0 = (float*)malloc(N * N * sizeof(float));
    float* h_c1 = (float*)malloc(N * N * sizeof(float));

    // Initialize input matrices with random values
    for (int i = 0; i < N * N; ++i) {
        h_a[i] = 1.0f; // Example initialization, you can use any desired initialization method
        h_b[i] = 1.0f; // Example initialization, you can use any desired initialization method
    }

    // Define the number of rows for each GPU
    int numRowsGPU0 = N / 2;
    int numRowsGPU1 = N - numRowsGPU0;

    // Start the timer for the entire process
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start, 0);

    // Launch matrix multiplication on GPU 0 for the first numRowsGPU0 rows
    matrixMultiplication(0, h_a, h_b, h_c0, numRowsGPU0);

    // Record end time for GPU 0
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    gpuTimes[0] = elapsedTime;

    // Reset the timer for GPU 1
    hipEventRecord(start, 0);

    // Launch matrix multiplication on GPU 1 for the remaining numRowsGPU1 rows
    matrixMultiplication(1, h_a + numRowsGPU0 * N, h_b, h_c1, numRowsGPU1);

    // Record end time for GPU 1
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    gpuTimes[1] = elapsedTime;

    // Calculate the total elapsed time by summing the GPU times
    float totalElapsedTime = gpuTimes[0] + gpuTimes[1];
    std::cout << "Total Elapsed Time: " << totalElapsedTime / 1000.0 << " seconds" << std::endl;

    // Combine h_c0 and h_c1 into a single result matrix h_c
    float* h_c = (float*)malloc(N * N * sizeof(float));

    // Copy h_c0 to the upper part of h_c
    for (int i = 0; i < numRowsGPU0; ++i) {
        memcpy(h_c + i * N, h_c0 + i * N, N * sizeof(float));
    }

    // Copy h_c1 to the lower part of h_c
    for (int i = 0; i < numRowsGPU1; ++i) {
        memcpy(h_c + (i + numRowsGPU0) * N, h_c1 + i * N, N * sizeof(float));
    }

    // Now, h_c contains the combined result
    printMatrix(h_c, N);

    // Free allocated memory
    free(h_a);
    free(h_b);
    free(h_c0);
    free(h_c1);
    free(h_c);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

