#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Function to initialize a matrix with random values
void initializeMatrix(float* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            //matrix[i * size + j] = static_cast<float>(rand()) / RAND_MAX;;
              matrix[i * size + j] =1.0f;
	}
    }
}

// Function to print a matrix (for debugging purposes)
void printMatrix(const float* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            std::cout << matrix[i * size + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    // Matrix size (assuming square matrices)
    const int matrixSize = 16384;  // Adjust the size based on your requirements

    // Allocate host memory for matrices
    float* h_A = new float[matrixSize * matrixSize];
    float* h_B = new float[matrixSize * matrixSize];
    float* h_C = new float[matrixSize * matrixSize];

    // Initialize matrices with random values
    initializeMatrix(h_A, matrixSize);
    initializeMatrix(h_B, matrixSize);

    // Allocate device memory for matrices
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, matrixSize * matrixSize * sizeof(float));
    hipMalloc((void**)&d_B, matrixSize * matrixSize * sizeof(float));
    hipMalloc((void**)&d_C, matrixSize * matrixSize * sizeof(float));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start the timer for memory copy (host to device)
    hipEventRecord(start, 0);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, matrixSize * matrixSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixSize * matrixSize * sizeof(float), hipMemcpyHostToDevice);

    // Stop the timer for memory copy (host to device)
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    
    float elapsedTimeMemCopyH2D;
    hipEventElapsedTime(&elapsedTimeMemCopyH2D, start, stop);
    std::cout << "Memory Copy (Host to Device) Time: " << elapsedTimeMemCopyH2D / 1000.0 << " seconds" << std::endl;

    // Initialize cuBLAS
    //they are used to encapsulate and manage resources efficiently.
   // They abstract away the internal details of the library and provide a clean interface for your application 
    //to interact with the GPU-related functionality.
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Start the timer for matrix multiplication
    hipEventRecord(start, 0);

    // Perform matrix multiplication using cuBLAS
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrixSize, matrixSize, matrixSize, &alpha, d_A, matrixSize, d_B, matrixSize, &beta, d_C, matrixSize);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time for matrix multiplication
    float elapsedTimeMultiplication;
    hipEventElapsedTime(&elapsedTimeMultiplication, start, stop);
    std::cout << "Matrix Multiplication Time (including Memory Copy): " << elapsedTimeMultiplication / 1000.0 << " seconds" << std::endl;

    
    hipEventRecord(start, 0);

    
    hipMemcpy(h_C, d_C, matrixSize * matrixSize * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    
    float elapsedTimeMemCopyD2H;
    hipEventElapsedTime(&elapsedTimeMemCopyD2H, start, stop);
    std::cout << "Memory Copy (Device to Host) Time: " << elapsedTimeMemCopyD2H / 1000.0 << " seconds" << std::endl;
    std::cout << "Total Time: " << elapsedTimeMemCopyH2D / 1000.0+ elapsedTimeMemCopyD2H / 1000.0+ elapsedTimeMultiplication / 1000.0<< " seconds" << std::endl;
   // printMatrix(h_C, matrixSize);
    
    
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}

